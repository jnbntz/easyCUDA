
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *sum, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
  {
      sum[i] = x[i] + y[i];
  }
}


int main(void)
{
  int N = 1<<20;
  float *x, *y, *sum;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  hipMallocManaged(&sum, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
    sum[i] = 0.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 256>>>(N, sum, x, y);
  add<<<1, 256>>>(N, sum, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(sum[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(sum);

  return 0;
}

